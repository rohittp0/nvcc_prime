#include "hip/hip_runtime.h"
#include <iostream>
#include <numeric>

__global__ void check_prime_cuda(const long long unsigned *number, bool *is_prime) {
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x + 2;

    if((*number) % i == 0)
        *is_prime = false;
}

__global__ void and_bools(const bool *in, bool *out) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (!in[i])
        *out = false;
}

bool check_prime(const long long unsigned number){
    unsigned long long root = (long long) sqrtl((long double)number),  *number_cuda;

    bool *is_prime = new bool(true);
    bool *results_cuda;

    hipMalloc((void **) &number_cuda, sizeof(long long unsigned));
    hipMalloc((void **) &results_cuda, sizeof(bool));

    hipMemcpy(number_cuda, &number, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(results_cuda, is_prime, sizeof(bool), hipMemcpyHostToDevice);

    check_prime_cuda<<<1, root>>>(number_cuda, results_cuda);

    hipMemcpy(is_prime, results_cuda, sizeof(bool), hipMemcpyDeviceToHost);


    hipFree(number_cuda);
    hipFree(results_cuda);

    return *is_prime;
}

int main(int argc, char **argv) {
    long long unsigned number;

    if(argc < 2)
        throw std::exception("Argument required: Number to check");
    else
        number = strtoull(argv[1], nullptr, 10);

//    for(long long unsigned i = 0; i < number; i++)
//        check_prime(i);

    bool is_prime = check_prime(number);

    if(argc > 2 && strcmp(argv[2],"--quite") == 0)
        return is_prime;

    std::cout << number << " is" << (is_prime ? " ": " not ") << "prime\n";

    return 0;
}
